#include "hip/hip_runtime.h"
// #CSCS CUDA Training 
//
// #Exercise 8_1 - texture memory, 2D stencil
//
// #Author Ugo Varetto
//
// #Goal: compare the performance of 2D stencil application with:
//        1) global memory
//        2) texture memory
//        3) shared memory 
//
// #Rationale: shows how texture memory is faster than global memory
//             when data are reused, thanks to (2D) caching; also
//             shows that for periodic boundary conditions using hw wrapping
//             is much faster than performing manual bounds checking
//
// #Solution: implement stencil computation accessing data in global, texture and shared memory
//
// #Code: 1) compute launch grid configuration
//        2) allocate data on host(cpu) and device(gpu)
//        3) map texture memory to pre-allocated gpu storage
//        4) copy data from host ro device
//        5) launch kernel
//        6) read data back
//        7) consume data (in this case print result)
//        8) release texture memory 
//        9) free memory
//        
// #Compilation: nvcc -arch=sm_13 8_1_texture-memory.cu -o texture-memory-2
//
// #Execution: ./texture-memory-2 
//
// #Note: the code is C++ also because the default compilation mode for CUDA is C++, all functions
//        are named with C++ convention and the syntax is checked by default against C++ grammar rules 
//
// #Note: -arch=sm_13 allows the code to run on every card available on Eiger and possibly even
//        on students' laptops; it's the identifier for the architecture before Fermi (sm_20)
// #Note: -arch=sm_13 is the lowest architecture version that supports double precision
//
// #Note: the example can be extended to read configuration data and array size from the command line
//        and could be timed to investigate how performance is dependent on single/double precision
//        and thread block size


#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <string>

typedef float real_t;

// read input data from global memory
__global__ void apply_stencil( const real_t* gridIn, 
                               const real_t* stencil,
                               real_t* gridOut,
                               int gridNumRows,
                               int gridNumColumns,
                               int stencilSize ) {
    // compute current thread id
    const int gridI = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridJ = blockIdx.x * blockDim.x + threadIdx.x;
    const int halfStencilSize = stencilSize / 2;
    if( gridI >= gridNumRows || gridJ >= gridNumColumns ) return;
    const int soff = halfStencilSize;
    real_t s = 0.f; 
    int si = 0;
    int sj = 0;
    for( int i = -halfStencilSize; i <= halfStencilSize; ++i) {
        si = gridI + i;
        if( si < 0 ) si += gridNumRows;
        else if( si >= gridNumRows ) si -= gridNumRows;
        for( int j = -halfStencilSize; j <= halfStencilSize; ++j ) {
            sj = gridJ + j;
            if( sj < 0 ) sj += gridNumColumns;
            else if( sj >= gridNumColumns ) sj -= gridNumColumns;
            s += gridIn[ si * gridNumColumns + sj ] * stencil[ ( i + soff ) * stencilSize + ( j + soff ) ];
        }
    }
    gridOut[ gridI * gridNumColumns + gridJ ] = s;
}

texture< real_t, 2 > gridInTex;
texture< real_t, 2 > stencilTex;


// read input data from global memory
__global__ void apply_stencil_texture( real_t* gridOut,
                                       int gridNumRows,
                                       int gridNumColumns,
                                       int stencilSize ) {
    // compute current thread id
    const int gridI = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridJ = blockIdx.x * blockDim.x + threadIdx.x;
    const int halfStencilSize = stencilSize / 2;
    if( gridI >= gridNumRows || gridJ >= gridNumColumns ) return;
    const int soff = halfStencilSize;
    real_t s = 0.f; 
    int si = 0;
    int sj = 0;
    for( int i = -halfStencilSize; i <= halfStencilSize; ++i) {
        si = gridI + i;
#ifndef TEXTURE_WRAP
        if( si < 0 ) si += gridNumRows;
        else if( si >= gridNumRows ) si -= gridNumRows;
#endif              
        for( int j = -halfStencilSize; j <= halfStencilSize; ++j ) {
             sj = gridJ + j;
#ifndef TEXTURE_WRAP
             if( sj < 0 ) sj += gridNumColumns;
             else if( sj >= gridNumColumns ) sj -= gridNumColumns;
#endif                               
             s += tex2D( gridInTex, sj, si ) * 
                  tex2D( stencilTex, j + soff, i + soff );
        }
    }
    gridOut[ gridI * gridNumColumns + gridJ ] = s;
}


void apply_stencil_ref( const real_t* gridIn,
                        const real_t* stencil,
                        real_t* gridOut,
                        int gridNumRows,
                        int gridNumColumns,
                        int stencilSize ) {
                                                    
     const int halfStencilSize = stencilSize / 2;
     const int soff = halfStencilSize;
     for( int r = 0; r != gridNumRows; ++r ) {
         for( int c = 0; c != gridNumColumns; ++c ) {
             real_t s = 0.f; 
             int si = 0;
             int sj = 0;
             for( int i = -halfStencilSize; i <= halfStencilSize; ++i) {
                 si = r + i;
                 if( si < 0 ) si += gridNumRows;
                 else if( si >= gridNumRows ) si -= gridNumRows;
                 for( int j = -halfStencilSize; j <= halfStencilSize; ++j ) {
                      sj = c + j;
                      if( sj < 0 ) sj += gridNumColumns;
                      else if( sj >= gridNumColumns ) sj -= gridNumColumns;
                     s += gridIn[ si * gridNumColumns + sj ] * stencil[ ( i + soff ) * stencilSize + ( j + soff ) ];
                 }
             }     
             gridOut[ r * gridNumColumns + c ] = s;
         }
     }
}

__global__ void init_grid( real_t* grid ) {
    const int gridI = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridJ = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = gridDim.x * blockDim.x;
    grid[ gridI * stride + gridJ ] = real_t( ( gridI + gridJ ) % 2 );                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                              
}


//------------------------------------------------------------------------------
int main( int , char**  ) {
    
    const int GRID_NUM_ROWS    = 0x100;// + 1; //257
    const int GRID_NUM_COLUMNS = 0x100;// + 1; //257
    const int GRID_SIZE = GRID_NUM_ROWS * GRID_NUM_COLUMNS;
    const int GRID_BYTE_SIZE = sizeof( real_t ) * GRID_SIZE;
    const int DEVICE_BLOCK_NUM_ROWS = 4; // num threads per row
    const int DEVICE_BLOCK_NUM_COLUMNS = 4; // num threads per columns
    const int STENCIL_EDGE_LENGTH = 3;
    const int STENCIL_SIZE = STENCIL_EDGE_LENGTH * STENCIL_EDGE_LENGTH;
    const int STENCIL_BYTE_SIZE = sizeof( real_t ) * STENCIL_SIZE;
    
    // block size: the number of threads per block multiplied by the number of blocks
    // must be at least equal to NUMBER_OF_THREADS 
    const int DEVICE_GRID_NUM_ROWS    = ( GRID_NUM_ROWS    + DEVICE_BLOCK_NUM_ROWS    - 1 ) / DEVICE_BLOCK_NUM_ROWS;
    const int DEVICE_GRID_NUM_COLUMNS = ( GRID_NUM_COLUMNS + DEVICE_BLOCK_NUM_COLUMNS - 1 ) / DEVICE_BLOCK_NUM_COLUMNS;
    // if number of threads is not evenly divisable by the number of threads per block 
    // we need an additional block; the above code can be rewritten as
    // if( NUMBER_OF_THREADS % THREADS_PER_BLOCK == 0) BLOCK_SIZE = NUMBER_OF_THREADS / THREADS_PER_BLOCK;
    // else BLOCK_SIZE = NUMBER_OF_THREADS / THREADS_PER_BLOCK + 1 
 
    //host allocated storage
    std::vector< real_t > host_stencil( STENCIL_SIZE, 1.0f / STENCIL_SIZE );
    std::vector< real_t > host_grid_in( GRID_SIZE );
    std::vector< real_t > host_grid_out( GRID_SIZE );

    // gpu allocated storage
    real_t* dev_grid_in  = 0;
    real_t* dev_grid_out = 0;
    real_t* dev_stencil  = 0;
    hipMalloc( &dev_grid_in,  GRID_BYTE_SIZE    );
    hipMalloc( &dev_grid_out, GRID_BYTE_SIZE   );
    hipMalloc( &dev_stencil,  STENCIL_BYTE_SIZE );
 
    // copy stencil to device
    hipMemcpy( dev_stencil, &host_stencil[ 0 ], STENCIL_BYTE_SIZE, hipMemcpyHostToDevice );

    init_grid<<< dim3( GRID_NUM_ROWS, GRID_NUM_COLUMNS, 1), dim3( 1, 1, 1 ) >>>( dev_grid_in );

    // copy initialized grid to host grid, faster than initializing on CPU
    hipMemcpy( &host_grid_in[ 0 ], dev_grid_in, GRID_BYTE_SIZE, hipMemcpyDeviceToHost );

    const dim3 blocks( DEVICE_GRID_NUM_COLUMNS, DEVICE_GRID_NUM_ROWS, 1 );
    const dim3 threads_per_block( DEVICE_BLOCK_NUM_COLUMNS, DEVICE_BLOCK_NUM_ROWS, 1 ); 

    //--------------------------------------------------------------------------
    // initialize events for timing execution
    hipEvent_t start = hipEvent_t();
    hipEvent_t stop = hipEvent_t();
    hipEventCreate( &start );
    hipEventCreate( &stop  );
    float e = 0.f;

    hipEventRecord( start );
    
    // execute kernel accessing global memory
    apply_stencil<<<blocks, threads_per_block>>>( dev_grid_in,
                                                  dev_stencil,
                                                  dev_grid_out,
                                                  GRID_NUM_ROWS,
                                                  GRID_NUM_COLUMNS,
                                                  STENCIL_EDGE_LENGTH );
    hipEventRecord( stop );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &e, start, stop );
    // read back result
    hipMemcpy( &host_grid_out[ 0 ], dev_grid_out, GRID_BYTE_SIZE, hipMemcpyDeviceToHost );
    // print grid
    std::cout << "Result: " << host_grid_out.front() << ".." << host_grid_out.back() << std::endl;
    std::cout << "Time:   " << e << " ms" << std::endl; 

    //--------------------------------------------------------------------------
    // describe data inside texture: 1-component floating point value in this case    
    const int BITS_PER_BYTE = 8;
    hipChannelFormatDesc cd = hipCreateChannelDesc( sizeof( real_t ) *  BITS_PER_BYTE,
                                                      0, 0, 0, hipChannelFormatKindFloat );
#ifdef TEXTURE_WRAP    
    gridInTex.addressMode[ 0 ] = hipAddressModeWrap;
    gridInTex.addressMode[ 1 ] = hipAddressModeWrap;
#endif                                                      
    // bind textures to pre-allocated storage
    int texturePitch = sizeof( real_t ) * GRID_NUM_COLUMNS;
    hipBindTexture2D( 0, &gridInTex,   dev_grid_in, &cd, GRID_NUM_COLUMNS, GRID_NUM_ROWS, texturePitch );
    texturePitch = sizeof( real_t ) * STENCIL_EDGE_LENGTH;
    hipBindTexture2D( 0, &stencilTex,  dev_stencil, &cd, STENCIL_EDGE_LENGTH, STENCIL_EDGE_LENGTH, texturePitch );                                                  

    hipEventRecord( start );

    // execute kernel accessing global memory
    apply_stencil_texture<<<blocks, threads_per_block>>>( dev_grid_out,
                                                          GRID_NUM_ROWS,
                                                          GRID_NUM_COLUMNS,
                                                          STENCIL_EDGE_LENGTH );
    
    hipEventRecord( stop );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &e, start, stop );
    // release texture
    hipUnbindTexture( &gridInTex  );
    hipUnbindTexture( &stencilTex );
    // read back result
    hipMemcpy( &host_grid_out[ 0 ], dev_grid_out, GRID_BYTE_SIZE, hipMemcpyDeviceToHost );
    // print grid
    std::cout << "Result: " << host_grid_out.front() << ".." << host_grid_out.back() << std::endl;
    std::cout << "Time:   " << e << " ms" << std::endl; 

    //--------------------------------------------------------------------------
    apply_stencil_ref( &host_grid_in[ 0 ],
                       &host_stencil[ 0 ],
                       &host_grid_out[ 0 ],
                       GRID_NUM_ROWS,
                       GRID_NUM_COLUMNS,
                       STENCIL_EDGE_LENGTH );
    std::cout << "Result: " << host_grid_out.front() << ".." << host_grid_out.back() << std::endl;

    

    // free memory
    hipFree( dev_grid_in );
    hipFree( dev_grid_out );
    hipFree( dev_stencil );

    return 0;
}
