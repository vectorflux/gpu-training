// #CSCS CUDA Training 
//
// #Exercise 7 - dot product with atomics - only works on Fermi and later
//
// #Author: Ugo Varetto
//
// #Goal: compute the dot product of two vectors performing all the computation on the GPU 
//
// #Rationale: shows how to perform the dot product of two vectors as a parallel reduction
//             with all the computation on the GPU; last step is done through synchronized
//             access to a shared variable.
// 
// #Solution: store scalar products in local cache and iterate over cache elements
//            performing incremental sums 
//
// #Code: 1) compute launch grid configuration
//        2) allocate data on host(cpu) and device(gpu)
//        3) initialize data directly on GPU
//        4) launch kernel
//        5) read data back 
//        6) free memory 
//             
// #Compilation: nvcc -arch=sm_20 4_parallel-dot-product-atomics.cu -o dot-product-atomics
//
// #Execution: ./dot-product-atomics
//
// #Note: as of CUDA 3.2 it seems that kernels do not stall anymore when invoking
//        __syncthreads from within an if block dependent on the thread id;
//       #see http://forums.nvidia.com/index.php?showtopic=178284
//
// #Note: the code is C++ also because the default compilation mode for CUDA is C++, all functions
//        are named with C++ convention and the syntax is checked by default against C++ grammar rules 
//
// #Note: -arch=sm_13 allows the code to run on every card available on Eiger and possibly even
//        on students' laptops; it's the identifier for the architecture before Fermi (sm_20)
//
// #Note: -arch=sm_13 is the lowest architecture version that supports double precision
//
// #Note: the example can be extended to read configuration data and matrix size from the command line

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <numeric>

typedef float real_t;

const size_t BLOCK_SIZE = 16;

__global__ void full_dot( const real_t* v1, const real_t* v2, real_t* out, int N ) {
    __shared__ real_t cache[ BLOCK_SIZE ];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    cache[ threadIdx.x ] = 0.f;
    while( i < N ) {
        cache[ threadIdx.x ] += v1[ i ] * v2[ i ];
        i += gridDim.x * blockDim.x;
    }    
    i = BLOCK_SIZE / 2;
    while( i > 0 ) {
        if( threadIdx.x < i ) cache[ threadIdx.x ] += cache[ threadIdx.x + i ];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }

    if( threadIdx.x == 0 ) atomicAdd( out, cache[ 0 ] );//   out[ blockIdx.x ] = cache[ 0 ];
}

real_t dot( const real_t* v1, const real_t* v2, int N ) {
    real_t s = 0;
    for( int i = 0; i != N; ++i ) {
        s += v1[ i ] * v2[ i ];
    }
    return s;
}


__global__ void init_vector( real_t* v, int N ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while( i < N ) {
        v[ i ] = 1.0f;//real_t( i ) / 1000000.f;
        i += gridDim.x * blockDim.x;
    } 
}


//------------------------------------------------------------------------------
int main(int argc, char** argv ) {
    
    const size_t ARRAY_SIZE = 1024;//1024 * 1024; //1Mi elements
    const int BLOCKS = 64;//512;
    const int THREADS_PER_BLOCK = BLOCK_SIZE;//256; // total threads = 512 x 256 = 128ki threads;                                   // each thread spans 8 array elements  
    const size_t SIZE = ARRAY_SIZE * sizeof( real_t );
    
    // device storage
    real_t* dev_v1 = 0;  // vector 1
    real_t* dev_v2 = 0;  // vector 2
    real_t* dev_out = 0; // result
    hipMalloc( &dev_v1,  SIZE );
    hipMalloc( &dev_v2,  SIZE );
    hipMalloc( &dev_out, sizeof( real_t ) );

    // host storage
    std::vector< real_t > host_v1( ARRAY_SIZE );
    std::vector< real_t > host_v2( ARRAY_SIZE );
    real_t host_out = 0.f;

    // initialize vector 1 with kernel; much faster than using for loops on the cpu
    init_vector<<< 1024, 256  >>>( dev_v1, ARRAY_SIZE );
    hipMemcpy( &host_v1[ 0 ], dev_v1, SIZE, hipMemcpyDeviceToHost );
    // initialize vector 2 with kernel; much faster than using for loops on the cpu
    init_vector<<< 1024, 256  >>>( dev_v2, ARRAY_SIZE );
    hipMemcpy( &host_v2[ 0 ], dev_v2, SIZE, hipMemcpyDeviceToHost );
    // initialize result
    hipMemset( dev_out, 0, sizeof( real_t) );   

    // execute kernel
    full_dot<<<BLOCKS, THREADS_PER_BLOCK>>>( dev_v1, dev_v2, dev_out, ARRAY_SIZE );
         
    // copy output data from device(gpu) to host(cpu)
    hipMemcpy( &host_out, dev_out, sizeof( real_t ), hipMemcpyDeviceToHost );

    // print dot product by summing up the partially reduced vectors
    std::cout << "GPU: " << host_out << std::endl;    

    // print dot product on cpu
    std::cout << "CPU: " << dot( &host_v1[ 0 ], &host_v2[ 0 ], ARRAY_SIZE ) << std::endl;

    // free memory
    hipFree( dev_v1 );
    hipFree( dev_v2 );
    hipFree( dev_out );

    return 0;
}
