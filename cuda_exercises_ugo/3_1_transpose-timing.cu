// #CSCS CUDA Training 
//
// #Example 3.1 - transpose matrix
//
// #Author: Ugo Varetto
//
// #Goal: compute the transpose of a matrix and time operation using
//        GPU's on-board performance counters through streams; print the result in ms (10^-3 s)    
//
// #Rationale: shows how to time GPU computation
//
// #Solution: straightworwad, simply compute the thread id associated with the element
//            and copy the transposed data into the output matrix; wrap kernel calls with event
//            recording and print time information         
//
// #Code: typical flow + timing: 
//        1) compute launch grid configuration
//        2) allocate data on host(cpu) and device(gpu)
//        3) initialize data directly on the GPU
//        4) create events
//        5) record start time
//        6) launch kernel
//        7) synchronize events to guarantee that kernel execution is finished
//        8) record stop time
//        9) read data back 
//        10) print timing information as stop - start time 
//        11) delete events 
//        12) free memory      
//        The code uses the default stream 0; streams are used to sychronize operations
//        to guarantee that all operations in the same stream are executed sequentially.
//                
// #Compilation: nvcc -arch=sm_13 3_1_transpose-timing.cu -o transpose-timing 
//
// #Execution: ./transpose-timing
//
// #Note: kernel invocations ( foo<<<...>>>(...) ) are *always* asynchronous and a call to 
//        cudaThreadSynchronize() is required to wait for the end of kernel execution from
//        a host thread; in case synchronous copy operations like cudaMemcpy(...,cudaDeviceToHost)
//        kernel execution is guaranteed to be terminated before data are copied
//
// #Note: the code is C++ also because the default compilation mode for CUDA is C++, all functions
//        are named with C++ convention and the syntax is checked by default against C++ grammar rules 
//
// #Note: -arch=sm_13 allows the code to run on every card with hw architecture GT200 (gtx 2xx) or better
//
// #Note: -arch=sm_13 is the lowest architecture version that supports double precision
//
// #Note: the example can be extended to read configuration data and matrix size from the command line

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

typedef float real_t;

__global__ void transpose( const real_t* in, real_t *out, int num_rows, int num_columns ) {
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int input_index = row * num_columns + col;
    const int output_index = col * num_rows + row; 
    out[ output_index ] = in[ input_index ];
}

__global__ void init_matrix( real_t* in ) {
    const int c = threadIdx.x + blockDim.x * blockIdx.x;
    const int r = threadIdx.y + blockDim.y * blockIdx.y;
    const int idx = c + gridDim.x * blockDim.x * r; 
    in[ idx ] = (real_t) idx; 
}

void print_matrix( const real_t* m, int r, int c, int stride ) {
    for( int i = 0; i != r; ++i ) {
        for( int j = 0; j != c; ++j ) std::cout << m[ i * stride + j ] << ' ';
        std::cout << '\n';
    }
    std::cout << std::endl;        
}

//------------------------------------------------------------------------------
int main(int argc, char** argv ) {
    
    const dim3 BLOCKS( 512, 512 );
    const dim3 THREADS_PER_BLOCK( 16, 16 ); 
    const int ROWS = 512 * 16; // 8192
    const int COLUMNS =  512 * 16; // 8192
    const size_t SIZE = ROWS * COLUMNS * sizeof( real_t );
    
    // device storage
    real_t* dev_in = 0;
    real_t* dev_out = 0;
    hipMalloc( &dev_in,  SIZE );
    hipMalloc( &dev_out, SIZE );
    
    // host storage
    std::vector< real_t > outmatrix( ROWS * COLUMNS );

    // initialize matrix with kernel; much faster than using
    // for loops on the cpu
    init_matrix<<<dim3( COLUMNS, ROWS ), 1>>>( dev_in );
    hipMemcpy( &outmatrix[ 0 ], dev_in, SIZE, hipMemcpyDeviceToHost );

    // print upper 4x4 left corner of input matrix
    std::cout << "INPUT MATRIX - " << ROWS << " rows, " << COLUMNS << " columns" << std::endl;
    print_matrix( &outmatrix[ 0 ], 4, 4, COLUMNS );
    
    // create events for timing execution
    hipEvent_t start = hipEvent_t();
    hipEvent_t stop  = hipEvent_t();
    hipEventCreate( &start );
    hipEventCreate( &stop  );
    
    // record time into start event 
    hipEventRecord( start, 0 ); // 0 is the default stream id
    // execute kernel
    transpose<<<BLOCKS, THREADS_PER_BLOCK>>>( dev_in, dev_out, ROWS, COLUMNS );
    
    // issue request to record time into stop event
    hipEventRecord( stop, 0 );
    // synchronize stop event to wait for end of kernel execution on stream 0
    hipEventSynchronize( stop );
    // compute elapsed time (done by CUDA run-time) 
    float elapsed = 0.f;
    hipEventElapsedTime( &elapsed, start, stop );
    
    std::cout << "Elapsed time (ms): " << elapsed << std::endl;

    // copy output data from device(gpu) to host(cpu)
    hipMemcpy( &outmatrix[ 0 ], dev_out, SIZE, hipMemcpyDeviceToHost );
    
    // print upper 4x4 corner of transposed matrix
    std::cout << "\nOUTPUT MATRIX - " << COLUMNS << " rows, " << ROWS << " columns" << std::endl;
    print_matrix( &outmatrix[ 0 ], 4, 4, ROWS );

    // free memory
    hipFree( dev_in );
    hipFree( dev_out );

    // release events
    hipEventDestroy( start );
    hipEventDestroy( stop  );


    return 0;
}
