#include "hip/hip_runtime.h"
// "mpi + cuda reduction" 

#ifdef GPU
#include <hip/hip_runtime.h>
#endif
#include <mpi.h>
#include <iostream>
#include <vector>
#include "mpierr.h"
#include <cmath>
#include <algorithm>
#include <sstream>

// compilation with mvapich2:
// nvcc -L/apps/eiger/mvapich2/1.6/mvapich2-gnu/lib -I/apps/eiger/mvapich2/1.6/mvapich2-gnu/include \
// -libumad -lmpich -lpthread -lrdmacm -libverbs -arch=sm_20 -DGPU \
// ~/projects/gpu-training/trunk/cuda_exercises_ugo/resources/mpiscratch/mpicuda2.cu


// note: using single precision floats because that's the only supported type
//       for atomics on CUDA 4

// note: experiment with different number of MPI tasks per GPU/node; using
//       256 Mi floats, 16 MPI tasks on two nodes (8 per node, 4 per GPUs)
//       CUDA fails to allocate memory exaclty for one task on each node;
//       Everything works fine with the same data with 8 tasks (4 per node, 2 per GPU ).

typedef float real_t;
#define MPI_REAL_T_ MPI_FLOAT

//------------------------------------------------------------------------------
#ifdef GPU
const int BLOCK_SIZE = 128;

__global__ void dot_product_kernel( const real_t* v1, const real_t* v2, int N, real_t* out ) {
    __shared__ real_t cache[ BLOCK_SIZE ];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i >= N ) return;
    cache[ threadIdx.x ] = 0.f;
    while( i < N ) {
        cache[ threadIdx.x ] += v1[ i ] * v2[ i ];
        i += gridDim.x * blockDim.x;
    }    
    i = BLOCK_SIZE / 2;
    while( i > 0 ) {
        if( threadIdx.x < i ) cache[ threadIdx.x ] += cache[ threadIdx.x + i ];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }
    if( threadIdx.x == 0 ) atomicAdd( out, cache[ 0 ] );   
}
#endif


//------------------------------------------------------------------------------
int main( int argc, char** argv ) {

    int numtasks = 0;
    int task     = 0;
    // INIT ENV
    MPI_( MPI_Errhandler_set( MPI_COMM_WORLD, MPI_ERRORS_RETURN ) );
    MPI_( MPI_Init( &argc, &argv ) );
    MPI_( MPI_Comm_size( MPI_COMM_WORLD, &numtasks ) );
    MPI_( MPI_Comm_rank( MPI_COMM_WORLD, &task  ) );
    std::vector< char > nodeid( MPI_MAX_PROCESSOR_NAME, '\0' );
    int len = 0;
    MPI_( MPI_Get_processor_name( &nodeid[ 0 ], &len ) );
    
    // RETRIEVE TOTAL NUMBER OF NODES USED, is there an easier way ?
    MPI_( MPI_Isend( &nodeid[ 0 ]...;
    if( task == 0 ) {
        typedef std::map< std::string, int > NodeCount;
        NodeCount ncount;
        std::vector< char > n( MPI_MAX_PROCESSOR_NAME, '\0' );
        for( int r = 0; r != numtasks; ++r ) {
            MPI_( MPI_Recv( &n[ 0 ],...;
            ++ncount[ &n[ 0 ] ];    
        }
        for( int r = 0; r != numtasks; ++r ) {
            MPI_( MPI_Isend( &nodecount...;
        }
    }
    int node_count = -1; 
    MPI_( MPI_Recv( &node_count...;
    const int tasks_per_node = numtasks / node_count;


    const int ARRAY_SIZE = 1024 * 1024 * 256; // 256 Mi floats x 2 == 2 GiB total storage
    // @WARNING: ARRAY_SIZE must be evenly divisible by the number of MPI processes
    const int PER_MPI_TASK_ARRAY_SIZE = ARRAY_SIZE / numtasks;
    if( ARRAY_SIZE % numtasks != 0  && task == 0 ) {
        std::cerr << ARRAY_SIZE << " must be evenly divisable by the number of mpi processes" << std::endl;
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    // PER TASK DATA INIT - in the real world this is the place where data are read from file
    // through the MPI_File_ functions or, less likely received from the root process
    std::vector< real_t > v1( ARRAY_SIZE / numtasks, 0. );
    std::vector< real_t > v2( ARRAY_SIZE / numtasks, 0. );
    for( int i = 0; i != PER_MPI_TASK_ARRAY_SIZE; ++i ) {
        v1[ i ] = 1;
        v2[ i ] = 1;  
    }

    real_t partial_dot = 0.;
#ifndef GPU
    for( int i = 0; i != PER_MPI_TASK_ARRAY_SIZE; ++i ) partial_dot += v1[ i ] * v2[ i ];
    std::ostringstream os;
    os << &nodeid[ 0 ] << " - rank: " << task << '\n';
    std::cout << os.str(); os.flush();     
#else
    // SELECT GPU = task % <num gpus on node>, note that with this
    // approach it is possible to support nodes with different numbers of GPUs
    int device_count = 0;
    if( hipGetDeviceCount( &device_count ) != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipGetDeviceCount FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    const int device =   ( task / tasks_per_node ) % device_count;
    std::ostringstream os;
    os << &nodeid[ 0 ] << " - rank: " << task << "\tGPU: " << device << '\n';
    std::cout << os.str(); os.flush();     

    if( hipSetDevice( device ) != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " cudaGetSetDevice FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    real_t* dev_v1   = 0;
    real_t* dev_v2   = 0;
    real_t* dev_dout = 0;
    if( hipMalloc( &dev_v1,   sizeof( real_t ) * PER_MPI_TASK_ARRAY_SIZE ) != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipMalloc FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    if( hipMalloc( &dev_v2,   sizeof( real_t ) * PER_MPI_TASK_ARRAY_SIZE ) != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipMalloc FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    if( hipMalloc( &dev_dout, sizeof( real_t ) * 1 ) != hipSuccess ) {
        std::cerr << task << ' ' << __LINE__ << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipMalloc FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    // MOVE DATA TO GPU
    if( hipMemcpy( dev_v1, &v1[ 0 ], sizeof( real_t ) * PER_MPI_TASK_ARRAY_SIZE,
                    hipMemcpyHostToDevice ) != hipSuccess ) {
        std::cerr << task << ' ' << __LINE__ << ' ' <<  hipGetErrorString( hipGetLastError() ) <<  " hipMemcpy FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;    
    }
    if( hipMemcpy( dev_v2, &v2[ 0 ], sizeof( real_t ) * PER_MPI_TASK_ARRAY_SIZE,
                    hipMemcpyHostToDevice ) != hipSuccess ) {
        std::cerr << task << ' ' << __LINE__ << ' ' <<  hipGetErrorString( hipGetLastError() ) <<  " hipMemcpy FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }     
    // INVOKE KERNEL
    const int NUM_THREADS_PER_BLOCK = BLOCK_SIZE; // must match size of buffer used for reduction
    const int NUM_BLOCKS = std::min( PER_MPI_TASK_ARRAY_SIZE  / NUM_THREADS_PER_BLOCK,
                                     0xffff ); // max number of blocks is 64k 
    // initialize partial dot product to zero
    if( hipMemset( dev_dout, 0, sizeof( real_t) ) != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipMemset FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    // actual on-device computation    
    dot_product_kernel<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>( dev_v1, dev_v2, PER_MPI_TASK_ARRAY_SIZE, dev_dout );
    // check for kernel launch errors: it is not possible to catch on-device execution errors but only
    // if there was an error launching the kernel
    if( hipGetLastError() != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " kernel launch FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;      
    }     

    // MOVE DATA TO CPU
    hipMemcpy( &partial_dot, dev_dout, sizeof( real_t ) * 1, hipMemcpyDeviceToHost );
#endif

    // REDUCE (SUM) ALL ranks -> rank 0
    real_t result = 0.;
    MPI_( MPI_Reduce( &partial_dot, &result, 1, MPI_REAL_T_, MPI_SUM, 0, MPI_COMM_WORLD ) );

    // IF RANK == 0 -> PRINT RESULT
    if( task == 0 ) {
        std::cout << "dot product result: " << result << std::endl;
    } 
  
#ifdef GPU
    // RELEASE GPU RESOURCES
    hipFree( dev_v1 );
    hipFree( dev_v2 );
    hipFree( dev_dout );
    hipDeviceReset(); 
#endif

    // RELEASE MPI RESOURCES   
    MPI_( MPI_Finalize() );

    return 0;
}
