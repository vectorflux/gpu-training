#include "hip/hip_runtime.h"
// ******* WORK IN PROGRESS ******* DOES NOT COMPILE


// #CSCS CUDA Training 
//
// #Exercise 5 - block matrix multiply
//
// #Author: Ugo Varetto


#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

typedef float real_t;

const size_t TILE_SIZE = 16;


__device__ real_t get_matrix_element( const real_t* m, 
                                      int blockCol,
                                      int blockRow,
                                      int col,
                                      int row,
                                      int num_columns ) {
                                      	
  
    return m[ ( blockRow * blockDim.y + row ) * num_columns + blockCol * blockDim.x + col ];

}

__global__ void matmul_coalesced( const real_t* m1, const real_t* m2, real_t* mout,
                        int m1_rows, int m1_columns, int m2_columns  ) { // m1_columns == m2_rows
                                                                         // mout = m1_rows x m2_columns
#ifdef AVOID_BANK_CONFLICTS	
	__shared__ real_t M1[ TILE_SIZE ][ TILE_SIZE ];
	__shared__ real_t M2[ TILE_SIZE ][ TILE_SIZE ];                   
#else
    __shared__ real_t M1[ TILE_SIZE ][ TILE_SIZE     ];
	__shared__ real_t M2[ TILE_SIZE ][ TILE_SIZE + 1 ];     
#endif		
	const int blockRow = blockIdx.y; 
    const int blockCol = blockIdx.x;
    const int row = threadIdx.y;
    const int col = threadIdx.x;
    real_t out = 0.f;
    for( int b = 0; b != gridDim.x; ++b ) {
    	//copy data into shared memory
    	M1[ row ][ col ] = get_matrix_element( m1, b, blockRow, row, col, m1_columns );
    	M2[ row ][ col ] = get_matrix_element( m2, blockCol, b, row, col, m2_columns );
        __syncthreads(); // required to guarantee that data are computed before next step
                         // where a thread accesses data computed by other threads
        for( int k = 0; k != TILE_SIZE; ++k ) {
            out += M1[ row ][ k ] * M2[ k ][ col ];       	
        }
        __synchthreads(); // required to avoid that some threads start modifying
                          // data in cache before all threads have exited for loop    
    }

   mout[ ( blockRow * blockDim.y + row ) * m2_columns + blockCol * blockDim.x + col ] = out;     
}

__global__ void matmul( const real_t* m1, const real_t* m2, real_t* mout,
                        int m1_rows, int m1_columns, int m2_columns  ) { // m1_columns == m2_rows
                                                                         // mout = m1_rows x m2_columns
	const int row = blockIdx.y * blockDim.y + threadIdx.x; 
    const int col = blockIdx.x * blockDim.x + threadIdx.y
    const int row = threadIdx.y;
    real_t out = m1[ row * m1_columns + 0 ] * m2[ 0 * m2_columns + col ];

    for( int k = 1; k != m1_columns; ++k ) {
    	out += m1[ row * m1_columns + k ] * m2[ k * m2_columns + col ];
    }

    mout[ row * m2_columns + col ] = out;
}

void matmul_ref( const real_t* m1, const real_t* m2, real_t* mout,
                 int m1_rows, int m1_columns, int m2_columns  ) {
                 	
    for( int row = 0; row != m1_rows; ++row ) {
    	for( int col = 0; col != m2_columns; ++col ) {
    		for( int k = 0; k != m1_columns; ++k ) {
    			mout[ row * m2_columns + col ] = m1[ row * m1_columns + k ] * m2[ k * m2_columns + col ];
    		}
    	}
    }


}